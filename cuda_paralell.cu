#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>
#include <string>
#include <fstream>

#define DEV_MATRIX(mat, x, y, matSize) mat[(x)*matSize + (y)]

using namespace std;

long long int timePassed(struct timespec start, struct timespec end)
{
    long long int seconds = (end.tv_sec - start.tv_sec);
    long long int nanos = (end.tv_nsec - start.tv_nsec);
    return seconds*1000000000 + nanos;
}

__device__ int* DevAllocateMatrix(int matSize)
{
    return (int*)malloc(sizeof(int)*matSize*matSize);
}

__host__ int** AllocateMatrix(int matSize)
{
    int** matrix = (int**)malloc(sizeof(int*)*matSize);
    for(int i = 0; i < matSize; i++)
        matrix[i] = (int*)malloc(sizeof(int)*matSize);
    return matrix;
}


void FreeMatrix(int** matrix, int matSize)
{
    for(int i = 0; i < matSize; i++)
        free(matrix[i]);
    free(matrix);
}

void CudaAllocateMatrix(int** ptr, int matSize)
{
    hipMalloc(ptr, sizeof(int)*matSize*matSize);
}

void CudaFreeMatrix(int* matrix)
{
    hipFree(matrix);
}

void CudaMatrixUpload(int** input, int* output, int matSize)
{
    for(int x = 0; x < matSize; x++)
    {
        hipMemcpy(output + (x*matSize), input[x], sizeof(int)*matSize, hipMemcpyHostToDevice);
    }       
}

//Pawan Harish and P. J. Narayanan 2007 "Accelerating large graph algorithms on the GPU using CUDA"
__global__ void FloydWarshall(int* graphMatrix, int* secondGraphMatrix, int matSize, int iteration)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index >= matSize*matSize)
    {
        return;
    }
    int first = index/matSize;
    int second = index%matSize;
    if(first >= second || first == iteration || second == iteration)
    {
        return;
    }
    int firstAlt = DEV_MATRIX(graphMatrix, first, iteration, matSize);
    int secondAlt = DEV_MATRIX(graphMatrix, iteration, second, matSize);
    int newDist = firstAlt + secondAlt;
    if(firstAlt != 0
    && secondAlt != 0
    && (newDist < DEV_MATRIX(graphMatrix, first, second, matSize) || DEV_MATRIX(graphMatrix, first, second, matSize) == 0))
    {
        DEV_MATRIX(secondGraphMatrix, first, second, matSize) = newDist;
        DEV_MATRIX(secondGraphMatrix, second, first, matSize) = newDist;
    }else
    {
        DEV_MATRIX(secondGraphMatrix, first, second, matSize) = DEV_MATRIX(graphMatrix, first, second, matSize);
        DEV_MATRIX(secondGraphMatrix, second, first, matSize) = DEV_MATRIX(graphMatrix, first, second, matSize);
    }
}

__global__ void CompressOutput(int* graphMatrix, int matSize, int* destinations, int* outputPtr, int outputSize)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index >= matSize*matSize)
    {
        return;
    }
    int first = index/matSize;
    int second = index%matSize;
    if(first == second)
    {
        return;
    }
    int firstNode = destinations[first];
    int secondNode = destinations[second];
    if(firstNode == 0 || secondNode == 0)
    {
        return;
    }
    firstNode--;
    secondNode--;
    int dist = DEV_MATRIX(graphMatrix, first, second, matSize);

    DEV_MATRIX(outputPtr, firstNode, secondNode, outputSize) = dist;
    DEV_MATRIX(outputPtr, secondNode, firstNode, outputSize) = dist;

}

__global__ void GeneratePermutations(int* permutations, int permutationLength, int permutationCount, int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index+offset >= permutationCount)
    {
        return;
    }

    int ogIndex = index;
    index += offset;

    int fact = 1;
    int lastMult;
    for(lastMult = 2; lastMult < permutationLength;)
    {
        fact *= lastMult;
        lastMult++;
    }
    lastMult--;

    for(int k = 0; k < permutationLength; k++)
    {
        DEV_MATRIX(permutations, ogIndex, k, permutationLength) = index/fact;
        index = index%fact;
        fact /= lastMult;
        lastMult--;
        if(lastMult < 1) lastMult = 1;
    }

    for(int k = permutationLength - 1; k > 0; k--)
    {
        for(int j = k-1; j>= 0; j--)
        {
            if(DEV_MATRIX(permutations, ogIndex, j, permutationLength) <= DEV_MATRIX(permutations, ogIndex, k, permutationLength))
            {
                DEV_MATRIX(permutations, ogIndex, k, permutationLength)++;
            }
        }
    }
}

__global__ void CalculateCycleLengths(int* graphMatrix, int* permutations, int matrixSize, int permutationCount, int* outputPtr, int offset, bool* isBest)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index+offset >= permutationCount)
    {
        return;
    }
    bool isValid = true;
    int sum = 0;
    int prev = 0;
    int daSize = matrixSize-1;
    for(int i = 0; i < daSize; i++)
    {
        int next = DEV_MATRIX(permutations, index, i, daSize)+1;
        if(DEV_MATRIX(graphMatrix, prev, next, matrixSize) == 0)
        {
            isValid = false;
        }
        sum += DEV_MATRIX(graphMatrix, prev, next, matrixSize);
        prev = next;
    }
    sum += DEV_MATRIX(graphMatrix, 0, prev, matrixSize);
    if(isValid)
    {
        outputPtr[index] = sum;
    }else{
        outputPtr[index] = 0;
    }
    isBest[index] = isValid;
}

__global__ void SelectSmallestResult(int* cycleLengths, bool* isShortest, int permLength)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index >= permLength)
    {
        return;
    }
    if(cycleLengths[index] == 0)
    {
        isShortest[index] = false;
        return;
    }
    int cycleLength = cycleLengths[index];
    for(int i = 0; i < permLength; i++)
    {
        if(cycleLengths[i] != 0 && (cycleLength > cycleLengths[i] || (cycleLength == cycleLengths[i] && index > i)))
        {
            isShortest[index] = false;
            return;
        }
    }
}

__global__ void ReturnSmallestResult(int* cycleLengths, bool* isShortest, int permLength, int* resultPtr)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index >= permLength)
    {
        return;
    }
    if(isShortest[index])
    {
        //printf("\nBest: %d %d\n", index, cycleLengths[index]);
        *resultPtr = cycleLengths[index];
    }
}

int Factorial(int val)
{
    int result = 1;
    for(int i = 2; i <= val; i++)
    {
        result *= i;
    }
    //printf("Factorial: %d %d\n", result, result*result);
    return result;
}


int main(int argc, char** argv)
{

    size_t real_size;
    int matSize;
    int destinationCount;

    int **graphMatrix;
    int *destinations;

    string line;
    ifstream inputf;

    if(argc < 2)
    {
        printf("Expected filepath to input data!\n");
        return 0;
    }
    inputf.open(argv[1]);
    if (!inputf.good())
    {
        printf("Failed to open file!\n");
        return 0;
    }

    getline(inputf, line);
    real_size = line.size();
    matSize = (int)real_size;
    destinationCount = 0;
    int matrixSize = matSize;
    graphMatrix = AllocateMatrix(matrixSize);
    destinations = (int*)malloc(sizeof(int)*matrixSize);

    for(int i = 0; i < matrixSize; i++)
        for(int y = 0; y < matrixSize; y++)
            graphMatrix[i][y] = 0;

    for(int i = 0; i < matrixSize; i++)
    {
        if(line[i] == '1')
        {
            destinationCount++;
            destinations[i] = destinationCount;
        }else
            destinations[i] = 0;
    }

    for(int point = 0; point < matrixSize; point++)
    {
        getline(inputf, line);
        real_size = line.size();
        matSize = (int)real_size;
        
        int connection = 0;
        for(int i = 0; i < matSize; i++)
        {
            if(line[i] != ' ')
            {
                graphMatrix[point][connection] *= 10;
                graphMatrix[point][connection] += (line[i] - '0');
            }else
            {
                graphMatrix[connection][point] = graphMatrix[point][connection];
                connection++;
            }
        }
    }


    inputf.close();
    for(int i = 0; i < matrixSize; i++)
    {
        for(int y = 0; y < matrixSize; y++)
        {
            //printf("%d, ", graphMatrix[i][y]);
        }
        //printf("\n");
    }

    int blockSize = 512;
    int blockCount;

    int result;
    int* devResult;
    struct timespec startTime, endTime;
    long long int timeElapsed;
    int* deviceMatrix;
    int* deviceMatrixCopy;
    int* transformedDeviceMatrix;
    int* deviceDestinations;

    hipMalloc(&deviceDestinations, sizeof(int)*matrixSize);
    hipMalloc(&devResult, sizeof(int));
    hipMemcpy(deviceDestinations, destinations, sizeof(int)*matrixSize, hipMemcpyHostToDevice);
    CudaAllocateMatrix(&deviceMatrix, matrixSize);
    CudaAllocateMatrix(&deviceMatrixCopy, matrixSize);
    CudaAllocateMatrix(&transformedDeviceMatrix, destinationCount);
    CudaMatrixUpload(graphMatrix, deviceMatrix, matrixSize);
    hipDeviceSynchronize();
    hipMemcpy(deviceMatrixCopy, deviceMatrix, sizeof(int)*matrixSize*matrixSize, hipMemcpyDeviceToDevice);

    FreeMatrix(graphMatrix, matrixSize);
    free(destinations);

    hipDeviceSynchronize();
    timespec_get(&startTime, TIME_UTC);

    blockCount = 1+((matrixSize*matrixSize)/blockSize);

    int* mem;
    for(int i = 0; i < matrixSize; i++)
    {
        FloydWarshall<<<blockCount, blockSize>>>(deviceMatrix, deviceMatrixCopy, matrixSize, i);
        hipDeviceSynchronize();
        
        mem = deviceMatrix;
        deviceMatrix = deviceMatrixCopy;
        deviceMatrixCopy = mem;
    }
    CompressOutput<<<blockCount, blockSize>>>(deviceMatrix, matrixSize, deviceDestinations, transformedDeviceMatrix, destinationCount);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    timespec_get(&endTime, TIME_UTC);
    timeElapsed = timePassed(startTime, endTime);
    //printf("[CUDA] Calculating distances time taken: %lld\n", timeElapsed);
    printf("%lld\n", timeElapsed);
    hipFree(deviceMatrix);

    int possiblePermutations = Factorial(destinationCount-1);
    int bestResult = 0;

    int permutationSpace = 40960;

    int permutationCount = permutationSpace/(destinationCount-1);
    blockCount = permutationCount/blockSize;
    if(permutationCount%blockSize != 0) blockCount++;

    int* devicePermutations;
    bool* deviceIsBestPermutation;
    int* devicePermutationLengths;
    hipMalloc(&devicePermutations, blockCount*blockSize*(destinationCount-1)*sizeof(int));
    hipMalloc(&deviceIsBestPermutation, blockCount*blockSize*sizeof(bool));
    hipMalloc(&devicePermutationLengths, blockCount*blockSize*sizeof(int));
    hipDeviceSynchronize();

    timespec_get(&startTime, TIME_UTC);
    
    for(int i = 0; i < possiblePermutations; i += blockCount*blockSize)
    {
        hipMemset(devicePermutations, 0, blockCount*blockSize*(destinationCount-1)*sizeof(int));
        hipDeviceSynchronize();

        GeneratePermutations<<<blockCount, blockSize>>>(devicePermutations, destinationCount-1, possiblePermutations, i);
        hipMemset(deviceIsBestPermutation, false, blockCount*blockSize*sizeof(bool));
        hipMemset(devicePermutationLengths, 0, blockCount*blockSize*sizeof(int));
        hipDeviceSynchronize();

        CalculateCycleLengths<<<blockCount, blockSize>>>(transformedDeviceMatrix, devicePermutations, destinationCount, possiblePermutations, devicePermutationLengths, i, deviceIsBestPermutation);
        hipDeviceSynchronize();

        SelectSmallestResult<<<blockCount, blockSize>>>(devicePermutationLengths, deviceIsBestPermutation, permutationCount);
        hipDeviceSynchronize();

        ReturnSmallestResult<<<blockCount, blockSize>>>(devicePermutationLengths, deviceIsBestPermutation, permutationCount, devResult);
        hipDeviceSynchronize();

        hipMemcpy(&result, devResult, sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        if(result < bestResult || bestResult == 0)
        {
            bestResult = result;
        }
    }

    timespec_get(&endTime, TIME_UTC);
    timeElapsed = timePassed(startTime, endTime);
    //printf("[CUDA] Brute force: %d time taken: %lld\n", bestResult, timeElapsed);
    printf("%lld\n", timeElapsed);

    hipFree(deviceIsBestPermutation);
    hipFree(devicePermutationLengths); 
    hipFree(transformedDeviceMatrix);

    return 0;
}
